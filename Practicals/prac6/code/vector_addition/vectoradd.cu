// In this assignment you will write a kernel for vector addition 
// you will also go through generalized processing from of a 
// GPU accelerated application. 
// These are:
//         1) initialize the host and data (allocate memory, load data, ...)
//         2) initialize the device (allocate memory, set its properties, ...)
//         3) transfer data to the device
//         4) run your kernel which will generate some result
//         5) transfer results to the host (eventually)
//         6) clean up (deallocate memory)
//         Run your code
//
// You should follow this assignment in steps mentioned in above list. 
// The TASK 1 correspond to initialization of the host, TASK 2 to 
// initialization of the device and so on.

// NOTE: You should finish your basic "Hello world" assignment first, before 
//       doing this one.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 4.0: Write your own kernel for vector addition
//
// To calculate the index of the data which given thread should operate
// on use pre-set variables threadIdx, blockIdx, blockDim and gridDim.
//
// Remember that kernel is written from point of view of a single thread,
// i.e. like serial code CPU.


// write your kernel here

//----------------------------------------------------------------------
__global__ void vector_add(int *d_C, int *d_A, int *d_B) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  d_C[index] = d_A[index] + d_B[index];
  // if (index % 524288 == 0) printf("Sanity check %d", d_C[index]);
}

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Our overall task is to calculate vector addition. To that end
  //         we have to declare arrays of float which will hold input data,
  //         vectors A and B and also the resulting vector C. All these 
  //         vectors will contain N elements (floats).
  // 
  // First you have to declare variables A, B and C. Remember that dynamically 
  // allocated arrays are expressed with pointers. Allocation of a pointer
  // looks like this: int *pointer_to_int;
  
  // Second step in initialization of the host is allocation of the memory
  // for our data. Allocation on the host could be done by using a 
  // function: void* malloc (size_t size);
  // pointer_to_int = (int*) malloc(size of the array in bytes);
  // The casting of the returned value is necessary because you want both 
  // sides of the expression of the same type. Since malloc returns void*,
  // which you can view as a pointer to a memory without any context, we
  // provide that context by telling the code that what this refers to is
  // actually an int.   
  
  // Last step is to initialize data on the host. We do not load any data
  // because we do not have any, which means you can initialize them to 
  // whatever value you want. However try to initialize them to values 
  // with which you can easily check that your implementation is correct.
  // However try to avoid using values which are same for every element.
  // You can initialize your data for example using a 'for' loop.

  size_t N = 8388608;
  
  // put your code here

  //----------------------------------------------------------------------
  int *h_A, *h_B, *h_C;

  h_A = (int*) malloc(N * sizeof(int));
  h_B = (int*) malloc(N * sizeof(int));
  h_C = (int*) malloc(N * sizeof(int));

  for (size_t f=0; f<N; f++) {
    h_A[f] = f + 1;
    h_B[f] = 2*f + 1;
    h_C[f] = 0;
  }

  //----------------------------------------------------------------------
  // TASK 2: In this task we initialize the GPU, declare variables which 
  //         resided on the GPU and then allocate memory for them.
  //           
  // We must start with device initialization. We do this by using same 
  // process we have used in our "Hello world" code.
  
  // Declaration of variables is no different than what we do for the host
  // it is the location to which the pointer points to which matters.
  
  // Lastly we allocate memory on the device by using cudaMalloc
  // cudaError_t cudaMalloc(void** pointer, size_t size);
  
  // put your code here
  
  //----------------------------------------------------------------------
  int deviceid = 0;
  int devCount;
  hipGetDeviceCount(&devCount);
  if(deviceid<devCount){
    hipSetDevice(deviceid);
  }
  else {
    printf("ERROR! Selected device is not available\n");
    return(1);
  }

  int *d_A, *d_B, *d_C;

  hipMalloc(&d_A, N * sizeof(int));
  hipMalloc(&d_B, N * sizeof(int));
  hipMalloc(&d_C, N * sizeof(int));
  //----------------------------------------------------------------------
  // TASK 3: Here we would like to copy the data from the host to the device
  
  // To do that we will use function 'cudaMemcpy'
  // cudaError_t cudaMemcpy(destination, source, size, direction);
  // where direction is either from the host to the device
  // 'cudaMemcpyHostToDevice' or from the device to the host 
  // 'cudaMemcpyDeviceToHost'.

  // put your code here

  //----------------------------------------------------------------------
  hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice);

  //----------------------------------------------------------------------
  // TASK 4.0: To write your vector addition kernel. Full task is above.
  //----------------------------------------------------------------------

  //----------------------------------------------------------------------
  // TASK 4.1: Now having data on the device and having a kernel for vector
  //           addition we would like to execute that kernel. 
  //
  // You can choose what ever grid configuration you desire, but take into 
  // account that, unless you have written the kernel otherwise, it cannot
  // handle data sizes which are not equal to 
  // (number of threads per block)*(number of blocks) == N !
  // In other words if N=200 and you are using 25 threads per block
  // you must launch your kernel with 8 blocks.
  
  // put your code here
  
  //----------------------------------------------------------------------
  dim3 Gd(256, 8, 8);
  dim3 Bd(8, 8, 8);
  vector_add<<<Gd, Bd>>>(d_C, d_A, d_B);
  //----------------------------------------------------------------------
  // TASK 5: Transfer data to the host.
  
  // put your code here

  //----------------------------------------------------------------------
  hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);
  
  if(N>1000){
	  printf("Check:\n");
	  for(int f=0; f<1000; f++){
		  printf("Is %d + %d = %d?\n", h_A[f], h_B[f], h_C[f]);
	  }
  }
  
  
  //----------------------------------------------------------------------
  // TASK 6: Free allocated resources.
  //
  // To do this on the device use cudaFree();
  
  // put your code here

  //----------------------------------------------------------------------
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
  // TASK 7: Run your code
  return(0);
}

